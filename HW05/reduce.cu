#include "hip/hip_runtime.h"
// implements the 'first add during global load' version (Kernel 4) for the parallel reduction
// g_idata is the array to be reduced, and is available on the device.
// g_odata is the array that the reduced results will be written to, and is available on the device.
// expects a 1D configuration.
// uses only dynamically allocated shared memory.
__global__ void reduce_kernel(float* g_idata, float* g_odata, unsigned int n)
{
    extern __shared__ float s[];
    int bid = blockIdx.x;
    int threads_per_block = blockDim.x;
    int tid = threadIdx.x;
    int thread = tid + threads_per_block * bid;

    // shared memory as input pointer
    float *s_input = &s[0];

    int g_input_index = tid + threads_per_block * bid * 2; // global input index
    int sec_g_input_index = g_input_index + threads_per_block; // second global input index

    // only pull into shared memory if within bounds of the inputted array
    if (g_input_index < N & sec_g_input_index < N)
    {
        s_input[thread] = g_odata[g_input_index] + g_odata[g_input_index + threads_per_block];
    } elif (g_input_index < N)
    {
        s_input[thread] = g_odata[g_input_index]
    }

    // wait for all threads to finish bringing in data
    __syncthreads();

    // sum the elements in the shared memory input array
    // might need boundary conditionals here
    for (int i = threads_per_block / 2; i > 0; i /= 2)
    {
        if (tid < i)
        {
            s_input[tid] += s_input[i + tid];
        }
        __syncthreads(); // since doing in place, have to wait for each round of threads to finish
    }

    // write shared summation to global output array
    if (thread == 0)
    {
        g_odata[bid] = s_input[0];
    }
}

// the sum of all elements in the *input array should be written to the first element of the *input array.
// calls reduce_kernel repeatedly if needed. _No part_ of the sum should be computed on host.
// *input is an array of length N in device memory.
// *output is an array of length = (number of blocks needed for the first call of the reduce_kernel) in device memory.
// configures the kernel calls using threads_per_block threads per block.
// the function should end in a call to hipDeviceSynchronize for timing purposes
__host__ void reduce(float** input, float** output, unsigned int N, unsigned int threads_per_block)
{
    int blocks_per_grid;
    int *orig_in = *input;
    int *temp;

    blocks_per_grid = N; // do this so first iteration of the loop is correct

    // iterates until the last run only used 1 block, means we are done
    while (blocks_per_grid != 1)
    {
        N = blocks_per_grid; // new size of input
        blocks_per_grid = N / (threads_per_block * 2); // new number of blocks needed

        // flip input and output pointers
        temp = *input;
        input = output;
        output = &temp;

        // call kernel with new values
        reduce_kernel<<<blocks_per_grid, threads_per_block, sizeof(float) * threads_per_block>>>(*input, *output, N);
    }
    // write final summation to first element of input array
    *orig_in[0] = *output[0];
}