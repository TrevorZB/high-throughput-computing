#include "hip/hip_runtime.h"
// Computes the convolution of image and mask, storing the result in output.
// Each thread should compute _one_ element of output.
// Shared memory should be allocated _dynamically_ only.
//
// image is an array of length n.
// mask is an array of length (2 * R + 1).
// output is an array of length n.
//
// Assumptions:
// - 1D configuration
// - blockDim.x >= 2 * R + 1
//
// The following should be stored/computed in shared memory:
// - The entire mask
// - The elements of image needed to compute the elements of output corresponding to the threads in the given block
// - The output image elements corresponding to the given block before it is written back to global memory
__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R)
{
    extern __shared__ float s[];
    int block_x = blockIdx.x;
    int thread_x = threadIdx.x;
    int thread = threadIdx.x + blockDim.x * block_x;
    int s_mask_size = 2 * R + 1;
    int s_image_size = 2 * R + block_x;

    // shared memory broken into mask, image, and output arrays
    float *s_mask = &s[0];
    float *s_image = &s[s_mask_size];
    float *s_output = &s[s_mask_size + s_image_size];

    // threads with id lower than the mask size
    // bring over their mask index from global to shared memory
    if (thread_x < s_mask_size - 1)
    {
        s_mask[thread_x] = mask[thread_x];
    }

    // calculates how many shared memory image indexes
    // this thread must bring over from global memory
    int work = (2 * R) / block_x + 1;

    // brings over this thread's designated image indexes
    // from global memory to shared memory
    for (int i = 0; i < work; i++)
    {
        int s_i_index = thread_x * work + i;
        int i_index = // index of the image that is brought over
        if (s_i_index < s_image_size - 1)
        {
            s_image[i_index] = image[i_index]; // ????? s_image index is probably supposed to be different than image index??
        }
    }

    // wait for threads in this block to finish bringing
    // over the mask and image values
    __syncthreads();

    // calculate the convolution summation
    float out = 0.0;
    for (int j = -R; j <= R; j++)
    {
        int image_val;
        int s_i_index = thread_x + j; // MIGHT NEED TO FIX THIS? SHOULD USE THREAD.X ID TO INDEX SHARED MEMORY IMAGE SINCE WE ONLY BROUGHT A SLICE OF THE IMAGE OVER
        if (s_i_index < 0 || s_i_index > n - 1)
        {
            image_val = 1; // out of bounds, default value
        } else
        {
            image_val = s_image[s_i_index]; // grab from shared memory
        }
        out += image_val * s_mask[j + R];
    }
    s_output[thread_x] = out;

    // write shared memory output to global memory output
}

// Makes one call to stencil_kernel with threads_per_block threads per block.
// The kernel call should be followed by a call to hipDeviceSynchronize for timing purposes.
//
// Assumptions:
// - threads_per_block >= 2 * R + 1
__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block)
{
    int shared_mem = 4 * R + 2 * threads_per_block + 1;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    stencil_kernel<<<blocks_per_grid, threads_per_block, sizeof(float) * shared_mem>>>(image, mask, output, n, R);
}