#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include "vscale.cuh"

void randomize_array(float *a, int size, float start, float stop)
{
    // randomize the seed, create distribution
    std::default_random_engine gen{static_cast<long unsigned int>(time(0))};
    std::uniform_real_distribution<float> dist(start, stop);

    for (int i = 0; i < size; i++)
    {
        a[i] = dist(gen);
    }
}

int main(int argc, char *argv[])
{
    int n = atoi(argv[1]);
    int threads_per_block = 512;
    int blocks = n / threads_per_block + 1;

    float *a_host = new float[n];
    float *a_dev;
    float *b_host = new float[n];
    float *b_dev;


    hipMalloc((void **)&a_dev, sizeof(float) * n);
    randomize_array(a_host, n, -10.0, 10.0);
    hipMemcpy(a_dev, a_host, sizeof(float) * n, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&b_dev, sizeof(float) * n);
    randomize_array(b_host, n, 0.0, 1.0);
    hipMemcpy(b_dev, b_host, sizeof(float) * n, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vscale<<<blocks, threads_per_block>>>(a_dev, b_dev, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(b_host, b_dev, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf("%f\n", ms);
    printf("%f\n", b_host[0]);
    printf("%f\n", b_host[n - 1]);

    delete[] a_host;
    delete[] b_host;
    hipFree(a_dev);
    hipFree(b_dev);
}